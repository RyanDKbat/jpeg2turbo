#include "hip/hip_runtime.h"

#include<stdio.h>
#include<math.h>
#include <time.h>
#include <stdlib.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#include "" 
#include<fstream>
#include "malloc.h"
#define SIZE 8
#define L 3
#define Lc 40
#define Es 1
#define Pi 3.14159265358979
#define Epsilon myexp(1)
#pragma warning(disable:4996)


double __device__  myexp(double x) {
    int i, k, m, t;
    int xm = (int)x;
    double sum;
    double e;
    double ef;
    double z;
    double sub = x - xm;
    m = 1;      //�׳��㷨��ĸ
    e = 1.0;  //e��xm
    ef = 1.0;
    t = 10;      //�㷨����
    z = 1;  //���ӳ�ʼ��
    sum = 1;
    //  printf("x=%f\n",x);
    //  printf("sub=%f\n",sub);
    if (xm < 0) {     //�ж�xm�Ƿ����0��
        xm = (-xm);
        for (k = 0; k < xm; k++) { ef *= 2.718281; }
        e /= ef;
    }
    else { for (k = 0; k < xm; k++) { e *= 2.718281; } }
    //  printf("e=%f\n",e);
    //  printf("xm=%d\n",xm);
    for (i = 1; i < t; i++) {
        m *= i;
        z *= sub;
        sum += z / m;
    }
    return sum * e;
}

double __device__    mk(double a, double s, double p, int c, int u)  //Mk(e)�ļ���//
{
    double mk;
    if (u == 1)
        mk = a - log(1 + myexp(a)) + 1 / 2 * s + 1 / 2 * p * (2 *double(c) - 1);
    else
        mk = -log(1 + myexp(a)) - 1 / 2 * s + 1 / 2 * p * (2 * double(c) - 1);
    return(mk);
}
 double __device__  abk(double t1, double k1, double t2, double k2)  //Ak(e)��Bk(e)�ļ���//
{
    double s1, s2, s;
    s1 = exp(t1 + k1); s2 = exp(t2 + k2);
    s = log(s1 + s2);
    return(s);
}
//����������//
 void __device__   DEC(double a[SIZE + 1], double ys[SIZE + 1], double yp[SIZE + 1], double e[SIZE])
{
    double me1[SIZE + 1], me2[SIZE + 1], me3[SIZE + 1], me4[SIZE + 1], me5[SIZE + 1], me6[SIZE + 1], me7[SIZE + 1], me8[SIZE + 1];
    double a0[SIZE], a1[SIZE], a2[SIZE], a3[SIZE], b0[SIZE + 1], b1[SIZE + 1], b2[SIZE + 1], b3[SIZE + 1];
    int i, u, c;
    for (i = 1; i <= SIZE; i++)
    {
        c = 0; u = 0;
        me1[i] = mk (a[i], ys[i], yp[i], c, u);
        c = 1; u = 1;
        me2[i] = mk (a[i], ys[i], yp[i], c, u);
        c = 0; u = 1;
        me3[i] = mk (a[i], ys[i], yp[i], c, u);
        c = 1; u = 0;
        me4[i] = mk  (a[i], ys[i], yp[i], c, u);
        c = 0; u = 0;
        me5[i] = mk  (a[i], ys[i], yp[i], c, u);
        c = 1; u = 1;
        me6[i] = mk  (a[i], ys[i], yp[i], c, u);
        c = 0; u = 1;
        me7[i] = mk  (a[i], ys[i], yp[i], c, u);
        c = 1; u = 0;
        me8[i] = mk  (a[i], ys[i], yp[i], c, u);
    }
    a0[0] = 1; a1[0] = 0; a2[0] = 0; a3[0] = 0;
    b0[SIZE] = 0; b1[SIZE] = 0; b2[SIZE] = 1; b3[SIZE] = 0;
    for (i = 1; i < SIZE; i++)
    {
        a0[i] = abk(a0[i - 1], me1[i], a2[i - 1], me6[i]);
        a1[i] = abk(a0[i - 1], me2[i], a2[i - 1], me5[i]);
        a2[i] = abk(a1[i - 1], me4[i], a3[i - 1], me7[i]);
        a3[i] = abk(a1[i - 1], me3[i], a3[i - 1], me8[i]);
    }
    for (i = SIZE - 1; i >= 1; i--)
    {
        b0[i] = abk(b0[i + 1], me1[i + 1], b1[i + 1], me2[i + 1]);
        b1[i] = abk(b3[i + 1], me3[i + 1], b2[i + 1], me4[i + 1]);
        b2[i] = abk(b1[i + 1], me5[i + 1], b0[i + 1], me6[i + 1]);
        b3[i] = abk(b2[i + 1], me7[i + 1], b3[i + 1], me8[i + 1]);
    }
    for (i = 1; i < SIZE; i++)
        e[i] = log(myexp(a0[i - 1] + 1 / 2 * yp[i] + b1[i]) + myexp(a1[i - 1] - 1 / 2 * yp[i] + b2[i]) + myexp(a2[i - 1] + 1 / 2 * yp[i] + b0[i]) + myexp(a3[i - 1] - 1 / 2 * yp[i] + b3[i])) - log(myexp(a0[i - 1] - 1 / 2 * yp[i] + b0[i]) + myexp(a1[i - 1] + 1 / 2 * yp[i] + b3[i]) + myexp(a2[i - 1] - 1 / 2 * yp[i] + b1[i]) + myexp(a3[i - 1] + 1 / 2 * yp[i] + b2[i]));
}
 void  __device__   interlace(double a[SIZE + 1], double b[SIZE + 1])  //��֯��//
{
    int i;
    for (i = 1; i < SIZE + 1; i++)    //����֯��
    {
        b[i] = a[SIZE + 1 - i];
    }
}
 void  __device__   uninterlace(double a[SIZE + 1], double b[SIZE + 1])  //�⽻֯��//
{
    int i;
    for (i = 1; i < SIZE + 1; i++)    //����⽻֯
    {
        b[i] = a[SIZE + 1 - i];
    }
}
__global__ void cudadecode(double *dataA, double *A)
{
    int x[SIZE + 1][SIZE + 1], y[SIZE + 1][3], y0[SIZE + 1], y1[SIZE + 1], y2[SIZE + 1];
    double y0_in[SIZE + 1], y00_in[SIZE + 1], y1_in[SIZE + 1], y2_in[SIZE + 1];
    double a[SIZE + 1], e[SIZE+1];
    double out1[SIZE + 1], out2[SIZE + 1];
    int i, j, k;
    int data1[16];
    int p = blockIdx.x * blockDim.x + threadIdx.x;

    if (p < 250000)//�߳���Լ�����ɸ�����������С����
    {
        k = 0;
        for (i = 15; i >= 0; i--)
        {
            data1[i] = int(dataA[p] - floor(dataA[p] / 10) * 10);
            dataA[p] /= 10;
        }
        for (i = 1; i <= SIZE; i++)
            for (j = 1; j < 3; j++)
            {
                x[i][j] = data1[k];
                k++;
                if (k == 15)
                    k = 0;
            }
        for (i = 1; i <= SIZE; i++)
            for (j = 1; j < 3; j++)
            {
                y[i][j] = double(2 * x[i][j] - 1);
            }
        for (i = 1; i <= SIZE; i++)   //����ת�����ŵ����Ŷȼ�Ȩ//
            for (j = 1; j < 3; j++)
            {
                if (j == 1)
                {
                    y0[i] = y[i][j];
                    y0_in[i] = Lc * double(y0[i]);
                }
                else
                {
                    if (i % 2 == 1)
                    {
                        y1[i] = y[i][j];
                        y1_in[i] = Lc * double(y1[i]);
                        y2[i] = 0;
                        y2_in[i] = Lc * double(y2[i]);
                    }
                    else
                    {
                        y1[i] = 0;
                        y1_in[i] = Lc * double(y1[i]);
                        y2[i] = y[i][j];
                        y2_in[i] = Lc * double(y2[i]);
                    }
                }
            }
        for (i = 1; i <= SIZE; i++)
            a[i] = 0;
        interlace(y0_in, y00_in);
        for (k = 1; k < 6; k++)    //����6��//
        {
            DEC(a, y0_in, y1_in, e);
            interlace(e, a);
            DEC(a, y00_in, y2_in, e);
            uninterlace(e, a);
        }
        DEC(a, y0_in, y1_in, e);
        interlace(e, a);
        DEC(a, y00_in, y2_in, e);
        for (i = 1; i <= SIZE; i++)
            out1[i] = a[i] + e[i] + y00_in[i];
        uninterlace(out1, out2);
        for (i = 1; i <= SIZE; i++)  //Ӳ�о����������������//
        {
            if (out2[i] >= 0)
                A[8 * p + i - 1] = 1;
            else
                A[8 * p + i - 1] = 0;
            ;
        }
    }
}

int main()
{

    int i, j, k;
    double* dataA = (double*)malloc(sizeof(double) * 250000);//����ɫ�ռ�
    double* dataB = (double*)malloc(sizeof(double) * 250000);
    double* dataC = (double*)malloc(sizeof(double) * 250000);
    double *A = (double*)malloc(sizeof(double) *  8 * 250000);//����ɫ��������ռ�
    double *B = (double*)malloc(sizeof(double) *  8 * 250000);
    double *C = (double*)malloc(sizeof(double) *  8 * 250000);
    double *d_dataA, *d_dataB, *d_dataC, *d_dataAA, *d_dataBB, *d_dataCC;
    hipMalloc((void**)&d_dataA, sizeof(double) * 250000);//����ɫ�Դ�ռ�
    hipMalloc((void**)&d_dataB, sizeof(double) * 250000);
    hipMalloc((void**)&d_dataC, sizeof(double) * 250000);
    hipMalloc((void**)&d_dataAA, sizeof(double)  * 8 * 250000);//����ɫ�Դ浼���ռ�
    hipMalloc((void**)&d_dataBB, sizeof(double)  * 8 * 250000);
    hipMalloc((void**)&d_dataCC, sizeof(double)  * 8 * 250000);
    memset(dataA, 0, sizeof(double) * 250000);
    memset(dataB, 0, sizeof(double) * 250000);
    memset(dataC, 0, sizeof(double) * 250000);
    memset(A, 0, sizeof(double) * 8*250000);
    memset(B, 0, sizeof(double) * 8*250000);
    memset(C, 0, sizeof(double) *8* 250000);
    hipMemset((void**)&d_dataA, 0, sizeof(double) * 250000);
    hipMemset((void**)&d_dataB, 0, sizeof(double) * 250000);
    hipMemset((void**)&d_dataC, 0, sizeof(double) * 250000);
    hipMemset((void**)&d_dataAA, 0, sizeof(double) *8* 250000);
    hipMemset((void**)&d_dataBB, 0, sizeof(double) *8* 250000);
    hipMemset((void**)&d_dataCC, 0, sizeof(double) *8* 250000);

   
    FILE* fw_red = fopen("/home/nvidia/project/transport/red_encode.txt", "r");
    for (i = 0; i < 250000; i++)
        fscanf(fw_red, "%lf", &dataA[i]);
    fclose(fw_red);          
    FILE* fw_green = fopen("/home/nvidia/project/transport/green_encode.txt", "r");
    for (j = 0; j < 250000; j++)
        fscanf(fw_green, "%lf", &dataB[j]);
    fclose(fw_green);
    FILE* fw_blue = fopen("/home/nvidia/project/transport/blue_encode.txt", "r");
    for (k = 0; k < 250000; k++)
        fscanf(fw_blue, "%lf", &dataC[k]);
    fclose(fw_blue);

    clock_t start = clock();
     hipMemcpy(d_dataA, dataA, sizeof(double) * 250000, hipMemcpyHostToDevice);//ԭ��ɫ���ݵ����Դ�
    hipMemcpy(d_dataAA, A, sizeof(double) * 8 * 250000, hipMemcpyHostToDevice);//��ɫ����ռ䵼���Դ�
    hipMemcpy(d_dataB, dataB, sizeof(double) * 250000, hipMemcpyHostToDevice);//��ɫ
    hipMemcpy(d_dataBB, B, sizeof(double) * 8 * 250000, hipMemcpyHostToDevice);
    hipMemcpy(d_dataC, dataC, sizeof(double) * 250000, hipMemcpyHostToDevice);//��ɫ
    hipMemcpy(d_dataCC, C, sizeof(double) * 8 * 250000, hipMemcpyHostToDevice);
    cudadecode << <256, 1024 >> > (d_dataA, d_dataAA);//cuda��������
    cudadecode << <256, 1024 >> > (d_dataB, d_dataBB);//cuda��������
    cudadecode << <256, 1024 >> > (d_dataC, d_dataCC);//cuda��������
    hipMemcpy(A, d_dataAA, sizeof(double)  * 8 * 250000, hipMemcpyDeviceToHost);//cuda�������������Դ浼���ڴ�
    hipMemcpy(B, d_dataBB, sizeof(double)* 8 * 250000, hipMemcpyDeviceToHost);
    hipMemcpy(C, d_dataCC, sizeof(double)  * 8 * 250000, hipMemcpyDeviceToHost);
    clock_t end = clock();
    double endtime = (double)(end - start) / CLOCKS_PER_SEC;
    cout << "totaltime:" << endtime * 1000 << "ms" << endl;
    ofstream out_red("/home/nvidia/project/transport/red_decode.txt");
    for (int i = 0; i < 250000 * 8; i++)//rgb�������
    {

        out_red << A[i];
        if ((i + 1) % 8 == 0)
        {
            out_red << "\n";
        }

    }
    out_red.close();
    ofstream out_green("/home/nvidia/project/transport/green_decode.txt");
    for (int j = 0; j < 250000 * 8; j++)//rgb�������
    {

        out_green << B[j];
        if ((j + 1) % 8 == 0)
        {
            out_green << "\n";
        }

    }
    out_green.close();
    ofstream out_blue("/home/nvidia/project/transport/blue_decode.txt");
    for (int k = 0; k < 250000 * 8; k++)//rgb�������
    {

        out_blue << C[k];
        if ((k + 1) % 8 == 0)
        {
            out_blue << "\n";
        }

    }
    
    free(dataA);
    free(dataB);
    free(dataC);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    hipFree(d_dataAA);
    hipFree(d_dataBB);
    hipFree(d_dataCC);
}

