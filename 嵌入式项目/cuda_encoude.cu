#include "hip/hip_runtime.h"

#include<stdio.h>
#include<math.h>
#include<fstream>
#define SIZE 8
#include "hip/hip_runtime.h"
#include "" 
#include "malloc.h"
#include <time.h>
#include <iostream>
#include"hip/device_functions.h"
using namespace std;
int __device__ RSC(int a, int* t1, int* t2)  //����������//
{
    int b, c;
    b = a ^ *t1 ^ *t2;
    c = b ^ *t2;
    *t2 = *t1;
    *t1 = b;
    return(c);
}
__global__ void cudaencode(int *dataA, int *A)
{
    int u[SIZE], u1[SIZE], c0[SIZE], c1[SIZE], c2[SIZE];
    int i, * p1, * p2, k, k1, k2, n;
    k1 = 0; k2 = 0; k = 0;
    p1 = &k1; p2 = &k2;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    if (j < 250000)//�߳���Լ�����ɸ�����������С����
        //for (j = 0; j < 250000; j++)//red�е�j������
        //{
    {
        for (i = 7; i >= 0; i--)
        {
            u[i] = dataA[j] % 10;
            dataA[j] /= 10;
        }                      //��λ
        for (i = 0; i < SIZE; i++)    //δ����֯����Ϣ���о��������������ϵͳ�����У�����//
        {
            c0[i] = u[i];
            c1[i] = RSC(u[i], p1, p2);
        }
        for (i = 0; i < SIZE; i++)    //����֯��
        {
            u1[i] = u[SIZE - 1 - i];
        }
        p1 = &k1; p2 = &k2;     //��λ�Ĵ�������//
        for (i = 0; i < SIZE; i++)    //��֯�����Ϣ���о��������������У�����//
            c2[i] = RSC(u1[i], p1, p2);
        for (i = 0; i < SIZE; i++)    //��ɾ����󸴽�//
            for (n = 0; n < 2; n++)
                if (n == 0)
                    A[j * 16 + i * 2 + n] = c0[i];
                else
                {
                    if (i % 2 == 0)
                        A[j * 16 + i * 2 + n] = c1[i];
                    else
                        A[j * 16 + i * 2 + n] = c2[i];
                }
    }
      
    //}


}
int main()
{

    int i, j, n;
    int pic_size;
    int* dataA = (int*)malloc(sizeof(int) * 250000);//����ɫ�ռ�
    int* dataB = (int*)malloc(sizeof(int) * 250000);
    int* dataC = (int*)malloc(sizeof(int) * 250000);
    int *A = (int*)malloc(sizeof(int) * 2 * 8 * 250000);//����ɫ��������ռ�
    int *B = (int*)malloc(sizeof(int) * 2 * 8 * 250000);
    int *C = (int*)malloc(sizeof(int) * 2 * 8 * 250000);
    int *d_dataA, *d_dataB, *d_dataC, * d_dataAA, * d_dataBB, * d_dataCC;
    hipMalloc((void**)&d_dataA, sizeof(int) * 250000);//����ɫ�Դ�ռ�
    hipMalloc((void**)&d_dataB, sizeof(int) * 250000);
    hipMalloc((void**)&d_dataC, sizeof(int) * 250000);
    hipMalloc((void**)&d_dataAA, sizeof(int) * 2 * 8 * 250000);//����ɫ�Դ浼���ռ�
    hipMalloc((void**)&d_dataBB, sizeof(int) * 2 * 8 * 250000);
    hipMalloc((void**)&d_dataCC, sizeof(int) * 2 * 8 * 250000);
    FILE* fw_pic = fopen("E:\\matin\\visual studio projects\\turbo_encode\\size.txt", "r");
    fscanf(fw_pic, "%d", &pic_size);    //��ȡͼƬ��С
    fclose(fw_pic);
    FILE* fw_red = fopen("E:\\matin\\visual studio projects\\turbo_encode\\red_data.txt", "r");
    for (j = 0; j < pic_size; j++)
        fscanf(fw_red, "%d", &dataA[j]);
    fclose(fw_red);           //��ȡred
    FILE* fw_green = fopen("E:\\matin\\visual studio projects\\turbo_encode\\green_data.txt", "r");
    for (j = 0; j < pic_size; j++)
        fscanf(fw_green, "%d", &dataB[j]);//��ȡgreen
    fclose(fw_green);
    FILE* fw_blue = fopen("E:\\matin\\visual studio projects\\turbo_encode\\blue_data.txt", "r");
    for (j = 0; j < pic_size; j++)
        fscanf(fw_blue, "%d", &dataC[j]);//��ȡblue
    fclose(fw_blue);
    hipMemcpy(d_dataA, dataA, sizeof(int) * 250000, hipMemcpyHostToDevice);//ԭ��ɫ���ݵ����Դ�
    hipMemcpy(d_dataAA, A, sizeof(int) * 2 * 8 * 250000, hipMemcpyHostToDevice);//��ɫ����ռ䵼���Դ�
    hipMemcpy(d_dataB, dataB, sizeof(int) * 250000, hipMemcpyHostToDevice);//��ɫ
    hipMemcpy(d_dataBB, B, sizeof(int) * 2 * 8 * 250000, hipMemcpyHostToDevice);
    hipMemcpy(d_dataC, dataC, sizeof(int) * 250000, hipMemcpyHostToDevice);//��ɫ
    hipMemcpy(d_dataCC, C, sizeof(int) * 2 * 8 * 250000, hipMemcpyHostToDevice);
    //dim3 dimgrid(32 * 32);
    //dim3 dimblock(64);
    clock_t start = clock();
    cudaencode << <256,1024 >> > (d_dataA, d_dataAA);//cuda��������
    cudaencode << <256, 1024 >> > (d_dataB, d_dataBB);
    cudaencode << <256, 1024 >> > (d_dataC, d_dataCC);
    hipMemcpy(A, d_dataAA, sizeof(int) * 2 * 8 * 250000, hipMemcpyDeviceToHost);//cuda�������������Դ浼���ڴ�
    hipMemcpy(B, d_dataBB, sizeof(int) * 2 * 8 * 250000, hipMemcpyDeviceToHost);
    hipMemcpy(C, d_dataCC, sizeof(int) * 2 * 8 * 250000, hipMemcpyDeviceToHost);
   clock_t end = clock();
    ofstream out_red("E:\\matin\\visual studio projects\\turbo_encode\\red_encode.txt");
    for (int k = 0; k < 250000 * 16; k++)//rgb�������
    {
     
            out_red << A[k];
            if((k+1)%16==0)
            {
                out_red << "\n";
            }
     
    }
    out_red.close();
    ofstream out_green("E:\\matin\\visual studio projects\\turbo_encode\\green_encode.txt");
    for (int k = 0; k < 250000 * 16; k++)//rgb�������
    {

        out_green << B[k];
        if ((k + 1) % 16 == 0)
        {
            out_green << "\n";
        }

    }
    out_green.close();
    ofstream out_blue("E:\\matin\\visual studio projects\\turbo_encode\\blue_encode.txt");
    for (int k = 0; k < 250000 * 16; k++)//rgb�������
    {

        out_blue << C[k];
        if ((k + 1) % 16 == 0)
        {
            out_blue << "\n";
        }

    }
    out_blue.close();
 
    free(dataA);
    free(dataB);
    free(dataC);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    hipFree(d_dataAA);
    hipFree(d_dataBB);
    hipFree(d_dataCC);
    double endtime = (double)(end - start) / CLOCKS_PER_SEC;
    cout << "totaltime:" << endtime * 1000 << "ms" << endl;
    return 0;
}
